﻿#include<hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <iostream>
#include <sstream>

__global__ void fillArray(double* array, int size, double value)
{ 
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	
	if(i<size){		
		array[i]=value;
	}
	
		

}


__global__ void applyDirichlet(double* load, double* csr_matrix, int* csr_col_device,int* csrRowPtr,int *isboundaryNode, int entries, int elementsX, int elementsY, int degree, double boundaryValue)
{ 
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int row=0;
	int col;
	int ucindex;
	int pointCount=(degree+1+(elementsX-1)*degree)*(degree+1+(elementsY-1)*degree)-1;
	double sum=0;
	double bound[4];
	bound[0]=1;
	bound[1]=0;
	bound[2]=0;
	bound[3]=0;
	//if (i>400)
		//printf("i_device: %i\n",i);
	if(i<pointCount+1){
		int start=csrRowPtr[i];
		int end =csrRowPtr[i+1];

		for(int j=start;j<end;j++)
		{	
			if(isboundaryNode[csr_col_device[j]]!=0)
			{					
				
				sum+=bound[isboundaryNode[csr_col_device[j]]-1]*csr_matrix[j];
			}
				
		}
		load[i]-=sum;

	}
	__syncthreads();		
	if(i<entries)
	{
		
		col=csr_col_device[i]; //spalte der großen matrix aber
		//printf("i: %i, isborder: %i  col: %i \n ", i,isboundaryNode[col],col);
		
		while((row<=pointCount) && (csrRowPtr[row]<=i))
		{		
			row++;
		}
		row--;
		
		
		//__syncthreads();		
			
		if((isboundaryNode[col]!=0)||(isboundaryNode[row]!=0)){			
			if(col!=row){				
				csr_matrix[i]=0;

			}
		}

		__syncthreads();		
		if((isboundaryNode[col]!=0)||(isboundaryNode[row]!=0)){
				if(col==row){
				//printf(" col : %i\n", col);
				csr_matrix[i]=1;	
				//load[col]=bound[isboundaryNode[col]-1];
				//printf(" Dirichlet auf : %i \n", col);
			}
			
			
		}

	// für jeden index i,jh in csr schaue ob i=j und i boundaryNode[i]=1 dann csr[i][j]=1 sonst wenn i!=j und i in boundary node dann csr[i][]=0
	}

		
}


__global__ void vectorDirichlet(double* load, double* csr_matrix, int* csr_col_device,int* csrRowPtr,int *isboundaryNode, int entries, int elementsX, int elementsY, int degree, double boundaryValue)
{ 
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int row=0;
	int col;
	int ucindex;
	int pointCount=(degree+1+(elementsX-1)*degree)*(degree+1+(elementsY-1)*degree)-1;
	double sum=0;
	double bound[4];
	bound[0]=1;
	bound[1]=0;
	bound[2]=0;
	bound[3]=0;
		
	if(i<entries)
	{
		
		col=csr_col_device[i]; //spalte der großen matrix aber

		
		while((row<=pointCount) && (csrRowPtr[row]<=i))
		{		
			row++;
		}
		row--;
		
	
		if((isboundaryNode[col]!=0)||(isboundaryNode[row]!=0)){
				if(col==row){
					
				load[col]=bound[isboundaryNode[col]-1];
			
			}
			
			
		}

	// für jeden index i,jh in csr schaue ob i=j und i boundaryNode[i]=1 dann csr[i][j]=1 sonst wenn i!=j und i in boundary node dann csr[i][]=0
	}

		
}
__global__ void BernBinomCoeff(double *M, int n)
{
	 int i= threadIdx.x;
	 int j= threadIdx.y;

	unsigned int top_0=1;
	unsigned int top_1=1;
	unsigned int bottom=1;
	unsigned int n_save=n;
	//guarantees that every step in the solution is smaller than the final solution thus avoiding overflow
	for (int d=1; d <= i; d++)
	{
		top_0*= n_save--;
        top_0 /= d;
    }

	n_save=n;
	
	for (int d=1; d <= j; d++)
	{
		top_1*= n_save--;
        top_1 /= d;
    }
	n_save=2*n;

	for (int d=1; d <= i+j; d++)
	{
		bottom*= n_save--;
        bottom /= d;
    }
	
	
	M[i+j*(n+1)]=(double)(top_0*top_1)/bottom;

	
		

	
}

__global__ void ass_A_exact(double a, double b,int *coo_row_device,int *coo_col_device, double*coo_value,int degree, int *elements, double *M, double *M_m, int elementsX, int elementsY)
{
	unsigned long int pointCount=(degree+1+(elementsX-1)*degree)*(degree+1+(elementsY-1)*degree);
	double *B;
	B=(double*)malloc((degree+1)*(degree+1)*(degree+1)*(degree+1)*sizeof(double));
	int i_glob;
	int j_glob;
	int shift;
	double sum=0;
	int element=threadIdx.x+blockIdx.x*blockDim.x;
	int n=degree;
	

	if(element<elementsX*elementsY)
	{
	 for (int i=0; i<=n;i++)
		for(int j=0; j<=n;j++)
			for (int k=0; k<=n;k++)
				for(int l=0; l<=n;l++)
				{
					sum=0;
					shift=i+j*(degree+1)+(degree+1)*(degree+1)*(k+l*(degree+1));
					
					if((i<n) && (k<n))
						sum+=M_m[i+n*k];
					
						if((i>0) && (i-1<n) && (k<n))
							sum-=M_m[i-1+n*k];
						if((k>0)&& (i<n) && (k-1<n))
							sum-=M_m[i+n*(k-1)];
						if((k>0) && (i>0) && (i-1<n)&& (k-1<n))
							sum+=M_m[i-1+n*(k-1)];
					
					
					B[shift]=M[j+l*(n+1)]*b/a*sum;
					sum=0;
					if((j<n) && (l<n))
						sum=M_m[j+n*l];
				
						if((j>0) && (j-1<n) && (l<n))
							sum-=M_m[j-1+n*l];
						if((l>0)&& (j<n) && (l-1<n))
							sum-=M_m[j+n*(l-1)];
						if((l>0) && (j>0) && (j-1<n)&& (l-1<n))
							sum+=M_m[j-1+n*(l-1)];
				
				
					B[shift]+=M[i+k*(n+1)]*a/b*(sum);
			
					B[shift]*=(double)(n*n)/(4*n*n-1);

					//if(((i+j+k==0) && (l<2)) || ((i+j+l==0) && (j<2)))
						B[shift]*=(double)(n*n)/(4*n*n-1);
					//else
						//B[shift]*=(double)(n*n)/(degree*(4*n*n-1));
					//start dumping values into coo list
			
						
						
				}
				for(int i=0; i<(n+1)*(n+1);i++)
				{
					for(int j=0; j<(n+1)*(n+1);j++)
					{
						i_glob=elements[element*(n+1)*(n+1)+i];
						j_glob=elements[element*(n+1)*(n+1)+j];
						
						coo_row_device[element*(n+1)*(n+1)*(n+1)*(n+1)+i+j*(n+1)*(n+1)]=i_glob;
						coo_col_device[element*(n+1)*(n+1)*(n+1)*(n+1)+i+j*(n+1)*(n+1)]=j_glob;
						
						//coo_row_device[element*(n+1)*(n+1)*(n+1)*(n+1)+i+j*(n+1)*(n+1)]=i_glob*pointCount+j_glob;
						coo_value[element*(n+1)*(n+1)*(n+1)*(n+1)+i+j*(n+1)*(n+1)]=B[i+j*(n+1)*(n+1)];
					}
				}

	}	
	free(B);
	
		
}

__global__ void reduce(double* data, int* index, int length)
{
	
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=0;
	if((index[i]==index[i+1]) && (index[i]!=index[i-1]))
		while(index[i]==index[i+j+1])
		{
			data[i]+=data[i+j+1];
			data[i+j+1]=0;
			index[i+j+1]=-1;
			j++;
		}



}

__global__ void split(int *index, int*cols, int*rows,int pointCount,int length)
{

	int i=threadIdx.x+blockIdx.x*blockDim.x;
	if(i<length)
	{
		rows[i]=index[i]/pointCount;
		
		
	}
	__syncthreads();
	if(i<length)
	{
		cols[i]=index[i]%pointCount;
		
	}
}