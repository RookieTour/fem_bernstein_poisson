﻿#include<hip/hip_runtime.h>



struct coordinates{         /* deklariert den Strukturtyp person */
    int i;
	int j;

  } ;
__global__ void BernBinomCoeff(double *M, int n)
{
	 int i= threadIdx.x;
	 int j= threadIdx.y;

	unsigned int top_0=1;
	unsigned int top_1=1;
	unsigned int bottom=1;
	unsigned int n_save=n;
	//guarantees that every step in the solution is smaller than the final solution thus avoiding overflow
	for (int d=1; d <= i; d++)
	{
		top_0*= n_save--;
        top_0 /= d;
    }

	n_save=n;
	
	for (int d=1; d <= j; d++)
	{
		top_1*= n_save--;
        top_1 /= d;
    }
	n_save=2*n;

	for (int d=1; d <= i+j; d++)
	{
		bottom*= n_save--;
        bottom /= d;
    }
	
	
	M[i+j*(n+1)]=(double)(top_0*top_1)/bottom;

	
}

__global__ void ass_A(double* A_device, int* g_mapping_device, int* c_mapping, double a, double b, int n, int Nx, int Ny, int color, int shiftx, int shifty)
{
//gaussian quadrature points
double q1=0.7886751346;
double q0=0.2113248654;

int iscolor;

//matrix for derivative of the form function [dX*[ξ,η,1]^t]_i=dx φ(ξ,η)_i and [dY*[ξ,η,1]^t]_i=dy φ(ξ,η)_i
/*next step precumpute dericative matrices for Bernstein polynomial k<=3 and sum over for loop in kernel*/

double dX[12]={0 , 1 , -1 ,
0 ,-1 , 1 ,
0 , 1 , 0,
0 ,-1 , 0};

double dY[12]={1, 0,-1,
-1, 0, 0,
1, 0, 0,
-1, 0, 1};

//allocate memory for element stiffness matrix on shared memory -> fast access
__shared__ double A_elem[16];

//global element indexes
int i= blockIdx.x;
int j= blockIdx.y;



//local element matrix indexes
int i_sub=threadIdx.x;
int j_sub=threadIdx.y;




iscolor=(color==c_mapping[i+Nx*j]);



if(iscolor){
//computing the summands for 2x2 gaussian quadrature
double B00=(dX[i_sub*3]*q0+dX[1+i_sub*3]*q0+dX[2+i_sub*3])*(dX[j_sub*3]*q0+dX[1+j_sub*3]*q0+dX[2+j_sub*3])/(a*a)
+(dY[i_sub*3]*q0+dY[1+i_sub*3]*q0+dY[2+i_sub*3])*(dY[j_sub*3]*q0+dY[1+j_sub*3]*q0+dY[2+j_sub*3])/(b*b);

double B01=(dX[i_sub*3]*q0+dX[1+i_sub*3]*q1+dX[2+i_sub*3])*(dX[j_sub*3]*q0+dX[1+j_sub*3]*q1+dX[2+j_sub*3])/(a*a)
+ (dY[i_sub*3]*q0+dY[1+i_sub*3]*q1+dY[2+i_sub*3])*(dY[j_sub*3]*q0+dY[1+j_sub*3]*q1+dY[2+j_sub*3])/(b*b);

double B10=(dX[i_sub*3]*q1+dX[1+i_sub*3]*q0+dX[2+i_sub*3])*(dX[j_sub*3]*q1+dX[1+j_sub*3]*q0+dX[2+j_sub*3])/(a*a)
+ (dY[i_sub*3]*q1+dY[1+i_sub*3]*q0+dY[2+i_sub*3])*(dY[j_sub*3]*q1+dY[1+j_sub*3]*q0+dY[2+j_sub*3])/(b*b);

double B11=(dX[i_sub*3]*q1+dX[1+i_sub*3]*q1+dX[2+i_sub*3])*(dX[j_sub*3]*q1+dX[1+j_sub*3]*q1+dX[2+j_sub*3])/(a*a)
+ (dY[i_sub*3]*q1+dY[1+i_sub*3]*q1+dY[2+i_sub*3])*(dY[j_sub*3]*q1+dY[1+j_sub*3]*q1+dY[2+j_sub*3])/(b*b);

}
__syncthreads();

if(iscolor){
int k=g_mapping_device[i_sub+(i+j*Nx)*4];
int l=g_mapping_device[j_sub+(i+j*Nx)*4];

//write in respective entries of global matrix
A_device[l+k*(Nx+1)*(Ny+1)]+=A_elem[i_sub+j_sub*4];
}

}

__global__ void ass_A_exact(double a, double b, coordinates *coo_index, double*coo_value,int degree, int *elements, double *M, double *M_m)
{
	double *B;
	B=(double*)malloc((degree+1)*(degree+1)*(degree+1)*(degree+1)*sizeof(double));
	int i_glob;
	int j_glob;
	int shift;
	double sum=0;
	int element=threadIdx.x;
	int n=degree;
	


	 for (int i=0; i<=n;i++)
		for(int j=0; j<=n;j++)
			for (int k=0; k<=n;k++)
				for(int l=0; l<=n;l++)
				{
					if(n>1)
					{
						sum=M_m[i+(n-1)*j];

						if(i>0)
							sum-=M_m[i-1+(n-1)*j];
						if(j>0)
							sum-=M_m[i+(n-1)*(j-1)];
						if((j>0) && (i>0))
							sum+=M_m[i-1+(n-1)*(j-1)];
					}	
					B[i+j+n*n*(k+l)]=M[i+j*n]*b/a*(sum);		
					sum=0;
					if(n>1)
					{
						sum=M_m[k+(n-1)*l];
				
						if(k>0)
							sum-=M_m[k-1+(n-1)*l];
						if(l>0)
							sum-=M_m[k+(n-1)*(l-1)];
						if((k>0) && (l>0))
							sum+=M_m[k-1+(n-1)*(l-1)];
					}
				
					B[i+j+n*n*(k+l)]+=M[k+l*n]*a/b*(sum);
					B[i+j+n*n*(k+l)]*=n*n/(4*n*n-1);
					//start dumping values into coo list
						shift=i+j+n*n*(k+l);
						
						i_glob=elements[k*(n+1)*(n+1)+i+j];
						j_glob=elements[k*(n+1)*(n+1)+k+l];
						
						coo_index[element*(n+1)*(n+1)+shift].i=i_glob;
						coo_index[element*(n+1)*(n+1)+shift].j=j_glob;
						coo_value[element*(n+1)*(n+1)+shift]=B[i+j+n*n*(k+l)];
				}

		
	free(B);
}