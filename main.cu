#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "helper_functions.h"  // helper for shared functions common to CUDA SDK samples
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"
//#include "hipsparse.h"
#include <hipsparse.h>
#include <hipblas.h>

#define _USE_MATH_DEFINES
#include <math.h>
using namespace std;

int checkcublasStatus ( hipblasStatus_t status, const char *msg ) 
{
    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        fprintf (stderr, "!!!! CUBLAS %s ERROR \n", msg);
        return 1;
    }
    return 0;
}

/* checkCusparseStatus: concise method for verifying cusparse return status */
int checkCusparseStatus ( hipsparseStatus_t status, const char *msg )
{
    if ( status != HIPSPARSE_STATUS_SUCCESS ) {
        fprintf (stderr, "!!!! CUSPARSE %s ERROR \n", msg);
        return 1;
    }
    return 0;
}

double* CGsolve(double *d_val, int* d_col, int* d_row, double* d_r, int nz, int N){
	
	const int max_iter =10000;
	int k;
	double tol=1e-15;
	double *d_x, *d_p, *d_Ax;
	double a, b, na, r0, r1,dot;
	double *x =new double[N];

	 for (int i = 0; i < N; i++) {
      
        x[i] =0;
    }

	  /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if ( checkcublasStatus (hipblasStatus_t, "!!!! CUBLAS initialization error\n") )
		printf("EXIT_FAILURE");

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if ( checkCusparseStatus (cusparseStatus, "!!!! CUSPARSE initialization error\n") ) printf("EXIT_FAILURE");

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr); 
    if ( checkCusparseStatus (cusparseStatus, "!!!! CUSPARSE hipsparseCreateMatDescr error\n") ) printf("EXIT_FAILURE");

    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
    
    checkCudaErrors( hipMalloc((void**)&d_x, N*sizeof(double)) );  
    //checkCudaErrors( hipMalloc((void**)&d_r, N*sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&d_p, N*sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&d_Ax, N*sizeof(double)) );



    hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);


    double alpha = 1.0;
    double alpham1 = -1.0;
    double beta = 0.0;
	r0 = 0.;

    cusparseStatus= hipsparseDcsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);
	
	hipblasDaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);

	
    hipblasStatus_t = hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
	
	
    k = 1;
    while (r1 > tol*tol && k <= max_iter) {
        if (k > 1) {
            b = r1 / r0;
            hipblasStatus_t = hipblasDscal(cublasHandle, N, &b, d_p, 1);
            hipblasStatus_t = hipblasDaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
        } else {
	    hipblasStatus_t = hipblasDcopy(cublasHandle, N, d_r, 1, d_p, 1);
        }

        hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax);
	


        hipblasStatus_t = hipblasDdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
	a = r1 / dot;
		
        hipblasStatus_t = hipblasDaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);

		
	na = -a;
        hipblasStatus_t = hipblasDaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);

        r0 = r1;
		
        hipblasStatus_t = hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    hipMemcpy(x, d_x, N*sizeof(double), hipMemcpyDeviceToHost);

 
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);

	return x;
	}

void printCSRMatrix(double *csr_values, int *csr_col, int *csr_row, int N){
		int k=0;
		int m, n;
		m=N;
		n=N;
		cout.precision(3);
	for(int j=0;j<m;j++){			
			for(int i=0;i<n;i++){
				if((i==csr_col[k]) &&(csr_row[j]<=k)){
					cout << csr_values[k] << "|";
					k++;
				}

				else
					cout << "0" << "|";
			}
			cout << endl;			
		}

	}


void quickSort(double *arr,int *index_i, int *index_j, int left, int right) {
      int i = left, j = right;
      double tmp;
	  int itemp, jtemp;
      int pivot = index_i[(left + right) / 2];

 
      /* partition */
      while (i <= j) {
            while (index_i[i] < pivot)
                  i++;
            while (index_i[j] > pivot)
                  j--;
            if (i <= j) {
                  tmp = arr[i];
				  itemp = index_i[i];
				  jtemp = index_j[i];

				  index_i[i]=index_i[j];
				  index_j[i]=index_j[j];
                  arr[i] = arr[j];
				  index_i[j]=itemp;
				  index_j[j]=jtemp;
                  arr[j] = tmp;
                  i++;
                  j--;
            }
      };
 
      /* recursion */
      if (left < j)
            quickSort(arr,index_i, index_j, left, j);
      if (i < right)
            quickSort(arr,index_i, index_j, i, right);
}


void SortCOO(double *coo_values, int *coo_row, int *coo_col,int PointsPerElement, int ElementCount)
{
	quickSort(coo_values,coo_row,coo_col,0,PointsPerElement*PointsPerElement*ElementCount-1);
		int start=0;
		int end;
		int j=0;
		for(int i=0;i<ElementCount*PointsPerElement;i++)
		{
			while(coo_row[j]==i)
			{
				j++;
			}
			end=j-1;
			quickSort(coo_values,coo_col,coo_row,start,end);
			start=end+1;
		}

}
//works as intended


int reduceCOO(double *coo_values, int *coo_row, int *coo_col,int PointsPerElement, int ElementCount)
{
	int zeroEntries=0;
	int j;
	for(unsigned long int i=0; i<PointsPerElement*PointsPerElement*ElementCount;i++)
	{
		j=1;
		while((coo_row[i]==coo_row[i+j]) && (coo_col[i]==coo_col[i+j]) && (i+j<PointsPerElement*PointsPerElement*ElementCount))
		{
			coo_values[i]+=coo_values[i+j];
			coo_values[i+j]=0;
			zeroEntries++;
			j++;
		}
			
		i+=j-1;		
	}

	
	//put zero entries at the end
	
	for(unsigned long int i=0; i<PointsPerElement*PointsPerElement*ElementCount;i++)
	{
		if(coo_values[i]==0)
		{
			j=0;
			while((coo_values[i+j]==0) && (i+j<PointsPerElement*PointsPerElement*ElementCount))
				j++;

			if(i+j<PointsPerElement*PointsPerElement*ElementCount)
			{
				coo_values[i]=coo_values[i+j];
				coo_values[i+j]=0;
				coo_row[i]=coo_row[i+j];
				coo_col[i]=coo_col[i+j];
			}
			//look for next nz entry j
			//copy entry to i, set j to zero

		}
		
	}
	return zeroEntries;
}

void convertCOOtoCSR(double *coo_values, int *coo_row, int *coo_col,int PointsPerElement, int ElementCount,int zeroEntries)
{
	double* CSR_values = new double[ElementCount*PointsPerElement*PointsPerElement-zeroEntries];
	int* CSR_index_col = new int[ElementCount*PointsPerElement*PointsPerElement-zeroEntries];
	int* CSR_pointer_row = new int[ElementCount*PointsPerElement];

	int k=0;
	int i=0;
	int j=0;
	while((i<ElementCount*PointsPerElement*PointsPerElement-zeroEntries)&& (j<ElementCount*PointsPerElement*PointsPerElement))
	{
		if (coo_values[j]!=0)
		{
			CSR_values[i]=coo_values[j];
			CSR_index_col[i]=coo_col[j];
			
			if(coo_row[j]==k)
			{
				CSR_pointer_row[k]=coo_row[j];
				k++;
			}
			i++;
			j++;
		}
		else
			j++;

	}
}

double func(double x, double y)
{
	return -2*M_PI*M_PI*cos(2*M_PI*x)*sin(M_PI*y)*sin(M_PI*y)-2*M_PI*M_PI*cos(2*M_PI*y)*sin(M_PI*x)*sin(M_PI*x);
}

//works as intended (correct for n=1 , n=2 tested only for specific cases n>2 "seems" okay)
int* createTriangulation(double *coordinatesX, double *coordinatesY, int degree, int elementsX, int elementsY, double sizeX, double sizeY)
{
	int ElementCount=elementsX*elementsY;
	int PointsPerElement=(degree+1)*(degree+1);
	int VertexPoints=(elementsX+1)*(elementsY+1);

	int *elements= new int[ElementCount*PointsPerElement];
	
	for(int k=0; k<ElementCount;k++)
	{
		//vertex nodes
		for(int i=0; i<2;i++)
			for(int j=0; j<2;j++)
			{
				elements[k*PointsPerElement+i+j*(degree+1)]=k+k/elementsX+i+j*(elementsX+1);
				coordinatesX[k*PointsPerElement+i+j*(degree+1)]=k*sizeX/elementsX+i*sizeX/elementsX;
				coordinatesY[k*PointsPerElement+i+j*(degree+1)]=k*sizeY/elementsY+j*sizeY/elementsY;
			}

		//center nodes
		for(int i=0; i<degree-1;i++)
			for(int j=0; j<degree-1;j++)
			{
				elements[k*PointsPerElement+i+2+(j+2)*(degree+1)]=VertexPoints+1+(degree-1)*elementsX
							+degree*k
								+(k/elementsX)*((degree*elementsX+1)*(degree-2)+(degree-1)*elementsX+1)
									+i+(degree*elementsX+1)*j;
				coordinatesX[k*PointsPerElement+i+2+(j+2)*(degree+1)]=0;//to be implemented
				coordinatesY[k*PointsPerElement+i+2+(j+2)*(degree+1)]=0;//to be implemented
			}

		//side nodes
							for(int i=0; i<degree-1;i++)
								for(int j=0; j<2;j++)
								{
										elements[k*PointsPerElement+i+2+j*(degree+1)]=VertexPoints+k*(degree-1)
											+(k/elementsX)*((degree*elementsX+1)*(degree-1))
												+i+j*((degree*elementsX+1)*(degree-1)+(degree-1)*elementsX);

										coordinatesX[k*PointsPerElement+i+2+j*(degree+1)]=0;//to be implemented
										coordinatesY[k*PointsPerElement+i+2+j*(degree+1)]=0;//to be implemented
								}
						for(int i=0; i<2;i++)
								for(int j=0; j<degree-1;j++)
								{
										elements[k*PointsPerElement+i+(j+2)*(degree+1)]=VertexPoints
											+(degree-1)*elementsX+k*(degree)+(k/elementsX)*((degree*elementsX+1)*(degree-2)+(degree-1)*elementsX+1)
												+i*degree+j*(degree*elementsX+1);

										coordinatesX[k*PointsPerElement+i+(j+2)*(degree+1)]=0;//to be implemented
										coordinatesY[k*PointsPerElement+i+(j+2)*(degree+1)]=0;//to be implemented
								}
									

	}
	return elements;
	
	

}

int* determineBorders(int elementsX, int elementsY, int degree)
{
	int pointCount=(degree+1+(elementsX-1)*degree)*(degree+1+(elementsY-1)*degree);
	
	int *boundaryNodes= new int[pointCount];

	for(int i=0; i<pointCount;i++)
		boundaryNodes[i]=0;

	/*			2		*/
	/*	-------------	*/
	/*	|			|	*/
	/* 	|			|	*/
	/*4 |			| 3	*/
	/*	|			|	*/
	/*	-------------	*/
	/*			1		*/



	//left border (3)
		//Vertex Nodes
			//i*(elementsX+1) (0<=i<=elementsY)
			for(int i=0; i<=elementsY;i++)
				boundaryNodes[i*(elementsX+1)]=3;
			
		//Side Nodes
			//(elementsX+1)*(elementsY+1) +elementsX*(degree-1)+i*(elementsX*(degree-1)+ elementsX+1)+ i/(degree-1)*(elementsX*(degree-1)) (0<=i<=(degree-1)*elementsY)
		if(degree>1){
			for(int i=0;i<=(degree-1)*elementsY;i++)
				boundaryNodes[(elementsX+1)*(elementsY+1) +elementsX*(degree-1)+i*(elementsX*(degree-1)+ elementsX+1)+ i/(degree-1)*(elementsX*(degree-1))]=3;			
		}
	//right border (4)
		//Vertex Nodes
			//i*(elementsX+1)+elementsX (0<=i<=elementsY)
			for(int i=0;i<=elementsY;i++)
				boundaryNodes[i*(elementsX+1)+elementsX]=4;
			
	//Side Nodes
			if(degree>1){
			//(elementsX+1)*(elementsY+1) +elementsX*(degree-1)+elementsX*degree+i*(elementsX*(degree-1)+ elementsX+1)+ i/(degree-1)*(elementsX*(degree-1)) (0<=i<=(degree-1)*elementsY)
				for(int i=0;i<(degree-1)*elementsY;i++)
					boundaryNodes[(elementsX+1)*(elementsY+1) +elementsX*(degree-1)+elementsX*degree+i*(elementsX*(degree-1)+ elementsX+1)+ i/(degree-1)*(elementsX*(degree-1))]=4;
			}

				//lower border (1)
		//Vertex Nodes
			//0 ... elementsX
			for(int i=0;i<=elementsX;i++)
				boundaryNodes[i]=1;
	
		//Side Nodes
			if(degree>1){
			//(elementsX+1)*(elementsY+1) ...(elementsX+1)*(elementsY+1) +elementsX*(degree-1)-1
				for(int i=(elementsX+1)*(elementsY+1);i<=(elementsX+1)*(elementsY+1) +elementsX*(degree-1)-1;i++)
					boundaryNodes[i]=1;
			}
	//upper border (2)
		//Vertex Nodes
			//(elementsX+1)*elementsY...(elementsX+1)*(elementsY+1)-1
			for(int i=(elementsX+1)*elementsY;i<=(elementsX+1)*(elementsY+1)-1;i++)
				boundaryNodes[i]=2;
		//Side Nodes
			if(degree>1){
				//pointCount -elementsX*(degree-1)...pointCount-1
			for(int i=pointCount -elementsX*(degree-1);i<=pointCount-1;i++)
				boundaryNodes[i]=2;
			}
			return boundaryNodes;
}
//has yet to be tested
double* assembleLoadVector(double a, double b, int degree, int *elements, int elementsX, int elementsY, double *nodes_x, double *nodes_y, int pointCount)
{
	int ElementCount=(elementsX+1)*(elementsY+1);
	int PointsPerElement=(degree+1)*(degree+1);
	int m;
	double xc,yc;	
	double *load_sub;
	load_sub=new double[PointsPerElement];
	double *load= new double[pointCount];
	for(int k=0;k<ElementCount;k++)
	{

		//get x,y cordinate from the element k of point P_0
		//cout << "Element k=" << k << endl;
		for(int i=0;i<PointsPerElement;i++)
		{
			xc=nodes_x[elements[PointsPerElement*k+i]];
			yc=nodes_y[elements[PointsPerElement*k+i]];
			//cout <<"i= " <<i << " global: " << elements[4*k+i]<< "  x: " << xc << "  y: "<< yc << endl;
			load_sub[i]=a*b/((degree+1)*(degree+1))*func(xc,yc);			
		}
			for(int i=0;i<4;i++){
				m=elements[i+k*PointsPerElement];
				load[m]+=load_sub[i];
			}
		
	}
	return load;
}



//works as intended maybe modify to overload and print int and doubles in one routine
void printMatrix(double* A, int n, int m)
{
	cout.precision(5);
	for(int j=0;j<m;j++){			
			for(int i=0;i<n;i++){			 
				cout << A[i+j*m]<< "|";
			}
			cout << endl;			
		}
		
}

//works as intended maybe modify to overload and print int and doubles in one routine
void printMatrix_int(int* A, int n, int m)
{
	cout.precision(2);
	for(int j=0;j<m;j++){			
			for(int i=0;i<n;i++){			 
				cout << A[i+j*m]<< "|";
			}
			cout << endl;			
		}
		
}

void testMatrixSym(double* A, int n, int m)
{
	
	for(int j=0;j<m;j++)
	{			
		for(int i=0;i<n;i++)
		{			 
			if (A[i+j*n]!=A[j+i*n])
			{
				cout << "Nicht symmetrisch" << endl;
				return;
			}
		
		}
		
	}
	cout <<"Ist symmetrisch"<< endl;		
}


 
int main()
{
	/*Simulation Variables*/
	int degree=1;	
	int elementsX=10;
	int elementsY=10;
	double sizeX=1.0;
	double sizeY=1.0;
	

	/*variables necessary for computation*/
	int ElementCount=elementsX*elementsY;
	int PointsPerElement=(degree+1)*(degree+1);
	int *elements=NULL;
	int *boundaryNodes=NULL;
	int pointCount=(degree+1+(elementsX-1)*degree)*(degree+1+(elementsY-1)*degree)-1;
	
	hipsparseHandle_t handle=0;
	hipsparseStatus_t status;
	double *LoadVector= new double[pointCount+1];

	/*allocation of necessary host memory*/
	double *coordinatesX= new double[ElementCount*PointsPerElement];
	double *coordinatesY= new double[ElementCount*PointsPerElement];	

	/*allocation of necessary device memory*/
	double	*coo_values_device;
	int		*coo_row_device;
	int		*coo_col_device;
	int		*elements_device;
	double	*M_device;
	double	*M_m_device;


	dim3 dimGrid(1+(elementsX*elementsY)/256,1,1);
	dim3 dimBlock(256,1,1);
	dim3 dimGridM(1,1,1);
	dim3 dimBlockM(degree+1,degree+1,1);
	dim3 dimBlockM_m(degree,degree,1);
	
	hipMalloc((void**)&coo_values_device, ElementCount*PointsPerElement*PointsPerElement*sizeof(double));
	hipMalloc((void**)&M_device, (degree+1)*(degree+1)*sizeof(double));
	hipMalloc((void**)&M_m_device, degree*degree*sizeof(double));
	hipMalloc((void**)&elements_device, ElementCount*PointsPerElement*sizeof(int));
	hipMalloc((void**)&coo_row_device, ElementCount*PointsPerElement*PointsPerElement*sizeof(int));
	hipMalloc((void**)&coo_col_device, ElementCount*PointsPerElement*PointsPerElement*sizeof(int));
	
	
	/*create triangulation for the simulation*/
	elements=createTriangulation(coordinatesX,coordinatesY,degree,elementsX,elementsY,sizeX,sizeY);
	boundaryNodes=determineBorders(elementsX, elementsY, degree);
	printf("boundary  nodes \n");
	printMatrix_int(boundaryNodes,8,8);
	/*copy necessarry memory to device*/	
	hipMemcpy(elements_device,elements, ElementCount*PointsPerElement*sizeof(int), hipMemcpyHostToDevice);


	
	/*assemble system matrix*/
	double a=sizeX/elementsX;
	double b=sizeY/elementsY;
	BernBinomCoeff<<<dimGridM, dimBlockM>>>(M_device, degree);
	BernBinomCoeff<<<dimGridM, dimBlockM_m>>>(M_m_device, degree-1);
	

	ass_A_exact<<<dimGrid, dimBlock>>>(a,b,coo_row_device, coo_col_device, coo_values_device,degree, elements_device, M_device, M_m_device,elementsX, elementsY);

	/* convert coo output into crs format*/
	//copy COO dataset from device to host
	double *coo_values = new double[ElementCount*PointsPerElement*PointsPerElement];
	int *coo_row = new int[ElementCount*PointsPerElement*PointsPerElement];
	int *coo_col = new int[ElementCount*PointsPerElement*PointsPerElement];

	hipMemcpy(coo_values,coo_values_device, ElementCount*PointsPerElement*PointsPerElement*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(coo_row,coo_row_device, ElementCount*PointsPerElement*PointsPerElement*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(coo_col,coo_col_device, ElementCount*PointsPerElement*PointsPerElement*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(coo_values_device);
	hipFree(coo_row_device);
	hipFree(coo_col_device);

	

	SortCOO(coo_values, coo_row, coo_col,PointsPerElement,ElementCount);
	int zeroEntries=reduceCOO(coo_values,coo_row,coo_col,PointsPerElement,ElementCount);

	for(int i=0; i<ElementCount*PointsPerElement*PointsPerElement;i++)
		printf("i: %i, col: %i\n",i, coo_col[i]);
	//allocating necessary sparse dataset memory

	int *csrRowPtr=0;
	double *csr_data_device;
	int *csr_col_device;
	
	hipMalloc((void**)&csr_col_device, (ElementCount*PointsPerElement*PointsPerElement-zeroEntries)*sizeof(int));
	hipMalloc((void**)&csrRowPtr, (pointCount+2)*sizeof(csrRowPtr[0]));
	hipMalloc((void**)&csr_data_device, (ElementCount*PointsPerElement*PointsPerElement-zeroEntries)*sizeof(double));
	hipMalloc((void**)&coo_row_device, (ElementCount*PointsPerElement*PointsPerElement-zeroEntries)*sizeof(int));
	
	//copy data back to device memory
	hipMemcpy(csr_data_device,coo_values, (ElementCount*PointsPerElement*PointsPerElement-zeroEntries)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(coo_row_device,coo_row, (ElementCount*PointsPerElement*PointsPerElement-zeroEntries)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(csr_col_device,coo_col, (ElementCount*PointsPerElement*PointsPerElement-zeroEntries)*sizeof(int), hipMemcpyHostToDevice);

	/*initialize cusparse library*/
	status=hipsparseCreate(&handle);
	if(status!=HIPSPARSE_STATUS_SUCCESS)
		cout << "CUSPARSE Library initialization failed" << endl;

	//convert to CSR format
	status=hipsparseXcoo2csr(handle, coo_row_device,ElementCount*PointsPerElement*PointsPerElement-zeroEntries,pointCount+1,csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
	if(status!=HIPSPARSE_STATUS_SUCCESS)
		cout << "Conversion from COO to CSR format failed" << endl;
	// csr_data_device , csrRowPtr, csr_col_device
	/*assemble load vector*/
	//assuming for the time beeing f=0

	 

	dim3 dimGridL(1+((pointCount+1)*(pointCount+1))/256,1,1);
	dim3 dimBlockL(256,1,1);
	double* LoadVector_device;

	hipMalloc((void**)&LoadVector_device,(pointCount+1)*sizeof(double));
	fillArray<<<dimGridL,dimBlockL>>>(LoadVector_device, pointCount+1, 0.0);
	

	
	/*apply dirichlet boundary conditions*/
	int *boundaryNodes_device;
	int nz=ElementCount*PointsPerElement*PointsPerElement-zeroEntries;


	hipMalloc((void**)&boundaryNodes_device,(pointCount+1)*sizeof(int));
	hipMemcpy(boundaryNodes_device,boundaryNodes,(pointCount+1)*sizeof(int),hipMemcpyHostToDevice);

	applyDirichlet<<<dimGridL,dimBlockL>>>(LoadVector_device, csr_data_device,csr_col_device,csrRowPtr,boundaryNodes_device, ElementCount*PointsPerElement*PointsPerElement-zeroEntries, elementsX, elementsY, degree, 2.0);
	
	int *row_index = new int[pointCount+2];
	int *col_index = new int[nz];
	
	hipMemcpy(coo_values,csr_data_device,(nz)*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(row_index,csrRowPtr,(pointCount+2)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(col_index,csr_col_device,(nz)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(LoadVector,LoadVector_device,(pointCount+1)*sizeof(double), hipMemcpyDeviceToHost);
	printf("nz: %i\n",nz);
	printf("pointCount: %i\n",pointCount+1);
	printf("values:\n");
	//printMatrix(coo_values,nz,1);
	//printCSRMatrix(coo_values,col_index,row_index,pointCount+1);
	//printf("row:\n");
	//printMatrix_int(row_index,pointCount+2,1);
	//printf("col:\n");
	//printMatrix_int(col_index,nz,1);
	printf("LoadVector:\n");
	//printMatrix(LoadVector,pointCount+1,1);
	//find upper boundary nodes, set them to v
	//find lower,left, right boundary nodes, set them to 0

	
	
	
	/*solve system of equations*/
	double *x=CGsolve(csr_data_device,csr_col_device,csrRowPtr, LoadVector_device,nz,pointCount+1);

	/*write solution into file*/


	


	

	/*for(int i=0; i<ElementCount*PointsPerElement*PointsPerElement; i++)
		cout << coo_row[i] << "  |  " << coo_col[i] << "  |  " << coo_values[i] << endl;
	
	
	
	/*free memory*/

	hipFree(elements_device);
	hipFree(M_device);
	hipFree(M_m_device);
	free(coo_values);


	/*
	//for(int i=1;i<16;i++)
	//runBernsteinSecondDegree(i,i);
	//runBernsteinSecondDegree(40	,40);
	//elements=createTriangulation(degree,elementsX,elementsY,1.0,1.0);
	//__global__ void ass_A_exact(a,b, coordinates *coo_index, double*coo_value,int degree, double *elements, double *M, double *M_m, degree);
	for(int k=0;k<elementsX*elementsY;k++){
		printMatrix(&coordinatesX[k*(degree+1)*(degree+1)],degree+1,degree+1);
		cout << endl;
	}*/
	
	

	stringstream fnAssembly;

	string Filename="";

	fnAssembly << "output" << elementsX <<"_" << elementsY<< ".txt";
	fnAssembly >> Filename;
	ofstream File(Filename);
	File << "{";
	for(int j=0;j<elementsY+1;j++)
	{
		File << "{";
		for(int i=0;i<elementsX;i++)		
			File <<x[i+j*(elementsX+1)] << "," ;
		File <<x[elementsX+j*(elementsX+1)];
		File << "}";
		if(j!=elementsY)
			File << ",";		
	}
	File << "}";
	File.close();
	

	double test;
	cin >>test;
    return 0;
}


